#include "hip/hip_runtime.h"
 #include <bits/stdc++.h>
#include "omp.h"
#include <sys/time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;
int DIM;
float epsilon;
int total_data_points;
float **input_set; //contains the input data points
float *input_set_gpu;
vector<int> vec_0_N;
int N,M,K;
float mincost;
vector<vector<float> > opti_centers;
float** final_centers;
vector<vector<int> > all_subset_points;
vector<vector<int> > k_indices_of_subset_points;


void print( vector<int> l){
    for(vector<int>::iterator it=l.begin(); it!=l.end() ; ++it)
            cout << " " << *it;
    cout<<endl;
}

void print( float *l){
    for(int i=0;i<DIM;i++)
        cout << l[i] << " ";
    cout << endl;
}
void print( vector<float> l){
    for(vector<float>::iterator it=l.begin(); it!=l.end() ; ++it)
            cout << " " << *it;
    cout<<endl;
}

float calc_distance(vector<float> &v1,vector<float> &v2)
{
	int sz = v1.size();
	float distsq = 0;
	for(int i=0;i<sz;i++)
	{
		distsq += ( (v1[i]-v2[i]) * (v1[i]-v2[i]) );
	}
	//cout << "distsq " <<  distsq <<  endl;
	return distsq;
}

float calc_distance(vector<float> &v1,float* v2)
{
	int sz = v1.size();
	float distsq = 0;
    for(int i=0;i<sz;i++)
	{
		distsq += ( (v1[i]-v2[i]) * (v1[i]-v2[i]) );
	}
	//cout << "distsq " <<  distsq <<  endl;
	return distsq;
}

__global__ void kernel(float *data, int row, int col,int dim) {

  printf("Element (%d, %d) = %d\n", row, col, data[(row*dim)+col]);

}
__global__ void gpu_fun(float *v1,float* v2,float* tmp_mindist,int row,int dim)
{
        int i =threadIdx.x;
        int j = row*dim +i;
        float val =  v1[i] - v2[j];
        //printf("inside kernel %d\n",v2[j]);
        //printf(" %d %d\n", i,val*val);// << i << endl;
		tmp_mindist[i] =  val*val;
}


vector<int> d2sampling(vector<vector<float> > &centers)
{
	map<pair<float,int>, int> mymap;
	vector<int> ans;
	for(int i=0;i<total_data_points;i++)
	{
		float mindist = INT_MAX;
        //  print(input_set[i]);
		for(int j=0;j<centers.size();j++)
		{
            float *centers1 = new float[DIM];
            for(int k=0;k<DIM;k++)
                centers1[k] = centers[j][k];
            float* centers_gpu;
            float* mindist_gpu;
            float * mindist_tmp= new float[DIM];
            *mindist_tmp = 0;
            hipMalloc(&mindist_gpu, sizeof(float)*DIM);
            hipMemcpy(mindist_gpu, mindist_tmp, sizeof(float)*DIM, hipMemcpyHostToDevice);

            const size_t a_size = sizeof(float) * DIM;
            hipMalloc(&centers_gpu, a_size);
            hipMemcpy(centers_gpu, centers1, a_size, hipMemcpyHostToDevice);

            //mindist = min(calc_distance(centers[j],input_set[i]),mindist );
            gpu_fun<<<1,DIM>>>(centers_gpu,input_set_gpu,mindist_gpu,i,DIM);
            hipMemcpy(mindist_tmp, mindist_gpu, sizeof(float)*DIM, hipMemcpyDeviceToHost);
            float res=0;
            for(int k=0;k<DIM;k++)
                res+=mindist_tmp[k];
            //cout << "res " <<  res << " mindist " << calc_distance(centers[j],input_set[i])<<endl;
            mindist = min(res,mindist);
            hipFree(centers_gpu);
            hipFree(mindist_gpu);

		}
		mymap.insert(make_pair(make_pair(mindist,i),0 ));
	}
	std::map<pair<float,int>, int>::reverse_iterator it;
	int count = 0;
	for (it=mymap.rbegin(); it!=mymap.rend(); ++it)
	{
		if(count >= N)
			break;
		ans.push_back(it->first.second);
		//cout << "dist " << it->first.first << endl;
		count++;
	}
	//cout << "finished" << endl;
	//print(ans);
	return ans;
}

__device__ static float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}


__global__ void cost(float* data,float *centers,float *ans,int dim,int num_centers) {

    //printf("%s\n", "here");
    int id = threadIdx.x;
    int blkid = blockIdx.x;
    float val=0;
    for(int i=0;i<dim;i++)
    {
         float tmp = data[id*dim+i]-centers[blkid*dim+i];
         val = val + tmp*tmp ;
    }
    atomicMin(&ans[id], val);
}

float cost(vector<vector<float> > &centers)
{
	float cst = 0;
	for(int i=0;i<total_data_points;i++)
	{
		float mindist = INT_MAX;
		for(int j=0;j<centers.size();j++)
		{
			mindist = min(calc_distance(centers[j],input_set[i]),mindist );
		}
		cst += mindist;
	}
	//cout << cst << " cost " << endl;
	return cst;
}


void generate_all_subsets(vector<int> data,int size, int left, int index)
{
	if(left==0){
			all_subset_points.push_back(data);
	}
    for(int i=index;i<size;i++){
    	data.push_back(vec_0_N[i]);
    	generate_all_subsets(data,size,left-1,i+1);
    	data.pop_back();
    }
}

void iterative_subset_open_mp(int i,int num_points)
{
	vector<int> tmp1;
	tmp1.push_back(i);
	//cout << i << " thread no. " << omp_get_thread_num() << endl;
	queue<vector<int> > k_indices_of_subset_points_t;
	k_indices_of_subset_points_t.push(tmp1);
	while((k_indices_of_subset_points_t.front()).size() < K)
	{
			vector<int> tmp = k_indices_of_subset_points_t.front();
			k_indices_of_subset_points_t.pop();
			//k_indices_of_subset_points.erase(k_indices_of_subset_points.begin());
			for(int j=0;j<num_points;j++)
			{
				tmp.push_back(j);
				k_indices_of_subset_points_t.push(tmp);
				tmp.pop_back();
			}
	}
	#pragma omp critical
	{
		while(!k_indices_of_subset_points_t.empty())
		{
			k_indices_of_subset_points.push_back(k_indices_of_subset_points_t.front());
			k_indices_of_subset_points_t.pop();
		}
	}
}

vector<int> generate_k_random(int n, int k)
{
	vector<int> v,v1;
	epsilon = 1/20;
	for(int i=0;i<n;i++)
		v1.push_back(i);
	for(int i=0;i<k;i++)
	{
		int randint = rand()%(n-i)+i;
		v.push_back(v1[randint]);
		iter_swap(v1.begin()+i,v1.begin()+randint);
	}
	return v;
}
int countn;
map<vector<vector<float> >,vector<int> > dp;
void main_openmp(vector<int> indices,vector<int> index_of_n_sampled_points)
{
    float *gpu_ans;
    hipMalloc((void**)&gpu_ans,sizeof(float)*total_data_points);
    float* dev_centers;
    hipMalloc((void **)&dev_centers, K*DIM* sizeof(float));
	vector<vector<float> > centers;
    float **center_new  = new float*[K];
    for(int i = 0;i < K; i++)
        center_new[i] = new float[DIM];
	vector<float> mean;
	vector<int> data = all_subset_points[indices[0]];
    float* init_ans = new float[total_data_points];
    for(int i = 0 ; i < total_data_points ; i++)
        init_ans[i] = INT_MAX;
    float* ans = new float[total_data_points];
	for(int i=0;i<DIM;i++)
		mean.push_back(0.0);
	int data_size = data.size();
	for(int i = 0 ; i < data_size;i++)
	{
		for(int j=0;j<DIM;j++)
		{
			mean[j]+= ((input_set[index_of_n_sampled_points[data[i]]][j] *1.0)/(data_size));
		}
	}
    for(int i=0;i<DIM;i++)
    {
        center_new[0][i] = mean[i];
    }
	centers.push_back(mean);
	for(int i=1;i<K;i++)
	{
		vector<float> mean1;
		vector<int> data1 = all_subset_points[indices[i]];
		for(int j=0;j<DIM;j++)
			mean1.push_back(0.0);
		int data_size1 = data1.size();
		map<vector<vector<float> >,vector<int> >::iterator it = dp.find(centers);
		if(it != dp.end())
		{
			index_of_n_sampled_points = it->second;
		}
		else
		{
			index_of_n_sampled_points = d2sampling(centers);
			#pragma omp critical
			{
				dp[centers] = index_of_n_sampled_points;
				//countn++;
				//cout << countn << endl;
			}
		}
		for(int j = 0 ; j < data_size1;j++)
		{
			for(int k=0;k<DIM;k++)
			{
				mean1[k]+= ((input_set[index_of_n_sampled_points[data1[j]]][k] *1.0)/(data_size1));
			}
		}
		centers.push_back(mean1);
        for(int x=0;x<DIM;x++)
        {
            center_new[i][x] = mean1[x];
        }
	}
    float new_cost = 0;

    for(int i = 0 ; i < K ; i++)
        hipMemcpy(dev_centers + i*DIM, center_new[i], DIM*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(gpu_ans,init_ans,sizeof(float)*total_data_points,hipMemcpyHostToDevice);
    cost<<<K,total_data_points>>>(input_set_gpu,dev_centers,gpu_ans,DIM,K);
    hipMemcpy( ans, gpu_ans, total_data_points* sizeof(float), hipMemcpyDeviceToHost );
    for(int i=0;i<total_data_points;i++)
    {
        new_cost += ans[i];
    }
    //cout << new_cost << "  " << cost(centers) << endl;
	#pragma omp critical
	{			//cout << "new cost " << centers.size() << endl;
		if(new_cost < mincost)
		{
			mincost = new_cost;
            for(int i=0;i<K;i++)
                for(int j=0;j<DIM;j++)
                    final_centers[i][j] = center_new[i][j];
			cout << mincost <<  " " << new_cost  << endl;
		}
	}
    hipFree(dev_centers);
    hipFree(gpu_ans);

}

int main()
{
	freopen("test1.txt","r", stdin);
	//ifstream myfile ("digitdata.txt");
	//int row = 1001,col = 157;
	int row = 12,col = 2;
    input_set = new float*[row];
    for(int i=0;i<row;i++)
        input_set[i] = new float[col];
	countn = 0;
	mincost = INT_MAX;
	K=3;
	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			int a1;
			cin >> a1;
            input_set[i][j] = a1;
			//v1.push_back((float)a1);
		}
		//input_set.push_back(v1);
	}
    hipMalloc((void **)&input_set_gpu, row*col* sizeof(float));
    for(int i = 0 ; i < row ; i++)    {
        // hipMalloc((void **)&hd_array[i], length[i] * sizeof(int));
        hipMemcpy(input_set_gpu + i*col, input_set[i], col*sizeof(float), hipMemcpyHostToDevice);
    }



	struct timeval start, end;
	gettimeofday(&start, NULL);
	//DIM = input_set[0].size();
    DIM = col;
	N= 10;
	M= 3;
    final_centers = new float*[K];
    for(int i = 0 ; i < K ; i++)
    {
        final_centers[i] = new float[DIM];
    }

    total_data_points = row;
	vector<int> index_of_n_sampled_points =  generate_k_random(total_data_points,N);
	//print(index_of_n_sampled_points);
	/*cout << v1.size() << endl;// " " << v1[0].size() << endl;
	for(int i =0;i<v1.size();i++)
		cout << v1[i] << " ";
	cout << endl;*/
	/*for(int i=0;i<row-1;i++)
	{
		for(int j=0;j<col;j++)
		{
			cout << i << " " << j << " " << input_set[i][j] << "\n " ;
		}
		cout << endl;
	}*/
	//return 0;
	for(int i=0;i<N;i++)
		vec_0_N.push_back(i);
	vector<int> data_subset;

	generate_all_subsets(data_subset,vec_0_N.size(),M,0);

	//generate_subset_indices(data_subset,vec_total_subset.size(),K,0);
	cout << "\n" << all_subset_points.size() << "\n";
	//iterative_subset();
	//cout << "\n" << k_indices_of_subset_points.size() << "\n";
	//int *a = new int[K];
	//for(int i=0;i<K;i++)
	//	a[i] = 0;
	//iterative_subset_1(a,all_subset_points.size(),0);
	int i1;
	#pragma omp parallel for num_threads(8)
	for(i1=0;i1<all_subset_points.size();i1++)
	{
		iterative_subset_open_mp(i1,all_subset_points.size());
	}

	cout << "Done" << endl;
	#pragma omp parallel for num_threads(8)
	for(i1=0;i1<k_indices_of_subset_points.size();i1++)
	{
		main_openmp(k_indices_of_subset_points[i1],index_of_n_sampled_points);
	}
	/*for(int i=0;i<k_indices_of_subset_points.size();i++)
	{
		for(int j=0;j<k_indices_of_subset_points[i].size();j++)
		{
			cout << k_indices_of_subset_points[i][j] << " " ;
		}
		cout << endl;
	}
	for(int i=0;i<K;i++)
	{
		cout << a[i] << " ";
	}
	cout << endl;*/
	//subsets(data_subset,centers,index_of_n_sampled_points,vec_0_N.size(),M,0);
	gettimeofday(&end, NULL);

	float delta = ((end.tv_sec  - start.tv_sec) * 1000000u +
         		end.tv_usec - start.tv_usec) / 1.e6;
    cout<<delta<<endl;
	cout << "final centers are" << endl;
	for(int i=0;i<K;i++)
	{
		print(final_centers[i]);
	}
    hipFree(input_set_gpu);

	return 0;
}
